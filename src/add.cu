
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to add the elements of two arrays on the GPU
__global__
void add(size_t n, float *x, float *y)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    size_t N = 1 << 20; // 1M elements

    // Allocate unified memory -- accessible from CPU or GPU
    float *x;
    float *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // Initialize x and y arrays on the host
    for (size_t i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (int(N) + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (size_t i = 0; i < N; i++) {
        maxError = std::fmax(maxError, std::fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
